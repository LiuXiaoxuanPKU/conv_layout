#include <hipDNN.h>
#include <iostream>

#define checkCUDNN(expression)  \
{                               \
	hipdnnStatus_t status = (expression); \
  if (status != HIPDNN_STATUS_SUCCESS) { \
    std::cerr << "Error on line " << __LINE__ << ": " \
              << hipdnnGetErrorString(status) << std::endl; \
    std::exit(EXIT_FAILURE); \
  } \
}

int main(int argc, char const *argv[]) {
  hipdnnHandle_t cudnn; // serve as a context object
  checkCUDNN(hipdnnCreate(&cudnn));

  const int height = 380;
  const int width = 380;

  hipdnnTensorDescriptor_t input_descriptor;
  checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
  checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor,
          /*format*/HIPDNN_TENSOR_NHWC,
          /*dataType*/HIPDNN_DATA_FLOAT,
          /*batch_size*/1,
          /*channels*/3,
          /*image_height*/height,
          /*image_width*/width));

  hipdnnTensorDescriptor_t output_descriptor;
  checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
  checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor,
          /*format*/HIPDNN_TENSOR_NHWC,
          /*dataType*/HIPDNN_DATA_FLOAT,
          /*batch_size*/1,
          /*channels*/3,
          /*image_height*/height,
          /*image_width*/width));

  hipdnnFilterDescriptor_t kernel_descriptor;
  checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor));
  checkCUDNN(hipdnnSetFilter4dDescriptor(kernel_descriptor,
          /*dataType*/HIPDNN_DATA_FLOAT,
          /*format*/HIPDNN_TENSOR_NCHW,
          /*out_channels*/3, 
          /*in_channels*/3,
          /*kernel_height*/3,
          /*kernel_width*/3));

  hipdnnConvolutionDescriptor_t convolution_descriptor;
  checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
  checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor,
                                           /*pad_height=*/1,
                                           /*pad_width=*/1,
                                           /*vertical_stride=*/1,
                                           /*horizontal_stride=*/1,
                                           /*dilation_height=*/1,
                                           /*dilation_width=*/1,
                                           /*mode=*/HIPDNN_CROSS_CORRELATION,
                                           /*computeType=*/HIPDNN_DATA_FLOAT));
  
  hipdnnConvolutionFwdAlgoPerf_t perf;
  int algo_cnt;
  hipdnnConvolutionFwdAlgo_t convolution_algorithm;
  checkCUDNN(
      requestedAlgoCount(cudnn,
            input_descriptor,
            kernel_descriptor,
            convolution_descriptor,
            output_descriptor,
            10, /*requestedAlgoCount*/
            &algo_cnt,
            &perf));

  convolution_algorithm = perf.algo;
  
  // In memory constrained environments, we may prefer HIPDNN_CONVOLUTION_FWD_SPECIFY_WORKSPACE_LIMIT
  
  size_t workspace_bytes = 0;
  checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
        input_descriptor,
        kernel_descriptor,
        convolution_descriptor,
        output_descriptor,
        convolution_algorithm,
        &workspace_bytes));

  std::cerr << "Workspace size: " << (workspace_bytes / 1048576.0) << "MB" << std::endl;
  
  // allocate memory
  void* d_workspace{nullptr};
  hipMalloc(&d_workspace, workspace_bytes);

  int image_bytes = 1 * 3 * height * width * sizeof(float);

  float array[1][height][width][3];
  float* d_input{nullptr};
  hipMalloc(&d_input, image_bytes);
  hipMemcpy(d_input, array, image_bytes, hipMemcpyHostToDevice);

  float* d_output{nullptr};
  hipMalloc(&d_output, image_bytes);
  hipMemset(d_output, 0, image_bytes);  

  const float kernel_template[3][3] = {
    {1, 1, 1},
    {1, -8, 1},
    {1, 1, 1}
  };

  float h_kernel[3][3][3][3];
  for (int kernel = 0; kernel < 3; ++kernel) {
    for (int channel = 0; channel <3; ++channel) {
      for (int row = 0; row < 3; ++row) {
        for (int column = 0; column < 3; ++column) {
          h_kernel[kernel][channel][row][column] = kernel_template[row][column];
        }
      }
    }
  }

  float* d_kernel{nullptr};
  hipMalloc(&d_kernel, sizeof(h_kernel));
  hipMemcpy(d_kernel, h_kernel, sizeof(h_kernel), hipMemcpyHostToDevice);


  float time;
  hipEvent_t start, stop;

  checkCUDNN( hipEventCreate(&start) );
  checkCUDNN( hipEventCreate(&stop) );
  checkCUDNN( hipEventRecord(start, 0) );

  int loop_times = 10000;
  for (int i = 0; i < loop_times; i++) {
    const float alpha = 1, beta = 0;
    checkCUDNN(hipdnnConvolutionForward(cudnn,
                                    &alpha,
                                    input_descriptor,
                                    d_input,
                                    kernel_descriptor,
                                    d_kernel,
                                    convolution_descriptor,
                                    convolution_algorithm,
                                    d_workspace,
                                    workspace_bytes,
                                    &beta,
                                    output_descriptor,
                                    d_output));
    float* h_output = new float[image_bytes];
  }

  checkCUDNN( hipEventRecord(stop, 0) );
  checkCUDNN( hipEventSynchronize(stop) );
  checkCUDNN( hipEventElapsedTime(&time, start, stop) );

  std::cout << "Run " << loop_times << " convolutions, run " << time << " ms";
  hipMemcpy(h_output, d_output, image_bytes, hipMemcpyDeviceToHost);
  // Do something with h_output ...

  delete[] h_output;
  hipFree(d_kernel);
  hipFree(d_input);
  hipFree(d_output);
  hipFree(d_workspace);

  hipdnnDestroyTensorDescriptor(input_descriptor);
  hipdnnDestroyTensorDescriptor(output_descriptor);
  hipdnnDestroyFilterDescriptor(kernel_descriptor);
  hipdnnDestroyConvolutionDescriptor(convolution_descriptor);

  hipdnnDestroy(cudnn);
}
