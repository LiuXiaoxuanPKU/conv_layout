
#include <hip/hip_runtime.h>
#include <cudnn.h>
#include <iostream>

#define checkCUDNN(expression)  \
{                               \
	cudnnStatus_t status = (expression); \
  if (status != CUDNN_STATUS_SUCCESS) { \
    std::cerr << "Error on line " << __LINE__ << ": " \
              << cudnnGetErrorString(status) << std::endl; \
    std::exit(EXIT_FAILURE); \
  } \
}

#define HANDLE_ERROR(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int main(int argc, char const *argv[]) {
  cudnnHandle_t cudnn; // serve as a context object
  checkCUDNN(cudnnCreate(&cudnn));

  const int height = 1000;
  const int width = 1000;

  cudnnTensorDescriptor_t input_descriptor;
  std::cout << "Before input descriptor" << std::endl;
  checkCUDNN(cudnnCreateTensorDescriptor(&input_descriptor));
  checkCUDNN(cudnnSetTensor4dDescriptor(input_descriptor,
          /*format*/CUDNN_TENSOR_NCHW,
          /*dataType*/CUDNN_DATA_FLOAT,
          /*batch_size*/1,
          /*channels*/3,
          /*image_height*/height,
          /*image_width*/width));
  std::cout << "Finish create input descriptor" << std::endl;
  cudnnTensorDescriptor_t output_descriptor;
  checkCUDNN(cudnnCreateTensorDescriptor(&output_descriptor));
  checkCUDNN(cudnnSetTensor4dDescriptor(output_descriptor,
          /*format*/CUDNN_TENSOR_NCHW,
          /*dataType*/CUDNN_DATA_FLOAT,
          /*batch_size*/1,
          /*channels*/3,
          /*image_height*/height,
          /*image_width*/width));

  std::cout << "Finish create output descriptor" << std::endl;
  cudnnFilterDescriptor_t kernel_descriptor;
  checkCUDNN(cudnnCreateFilterDescriptor(&kernel_descriptor));
  checkCUDNN(cudnnSetFilter4dDescriptor(kernel_descriptor,
          /*dataType*/CUDNN_DATA_FLOAT,
          /*format*/CUDNN_TENSOR_NCHW,
          /*out_channels*/3, 
          /*in_channels*/3,
          /*kernel_height*/3,
          /*kernel_width*/3));

  std::cout << "Finish create filter descriptor" << std::endl;
  cudnnConvolutionDescriptor_t convolution_descriptor;
  checkCUDNN(cudnnCreateConvolutionDescriptor(&convolution_descriptor));
  checkCUDNN(cudnnSetConvolution2dDescriptor(convolution_descriptor,
                                           /*pad_height=*/1,
                                           /*pad_width=*/1,
                                           /*vertical_stride=*/1,
                                           /*horizontal_stride=*/1,
                                           /*dilation_height=*/1,
                                           /*dilation_width=*/1,
                                           /*mode=*/CUDNN_CROSS_CORRELATION,
                                           /*computeType=*/CUDNN_DATA_FLOAT));
  
  std::cout << "Finish create conv descriptor" << std::endl;
  cudnnConvolutionFwdAlgoPerf_t perf;
  int algo_cnt;
  cudnnConvolutionFwdAlgo_t convolution_algorithm;
  checkCUDNN(
      cudnnFindConvolutionForwardAlgorithm(cudnn,
            input_descriptor,
            kernel_descriptor,
            convolution_descriptor,
            output_descriptor,
            10, /*requestedAlgoCount*/
            &algo_cnt,
            &perf));

  convolution_algorithm = perf.algo;
  
  // In memory constrained environments, we may prefer CUDNN_CONVOLUTION_FWD_SPECIFY_WORKSPACE_LIMIT
  
  size_t workspace_bytes = 0;
  checkCUDNN(cudnnGetConvolutionForwardWorkspaceSize(cudnn,
        input_descriptor,
        kernel_descriptor,
        convolution_descriptor,
        output_descriptor,
        convolution_algorithm,
        &workspace_bytes));

  std::cerr << "Workspace size: " << (workspace_bytes / 1048576.0) << "MB" << std::endl;
  
  // allocate memory
  void* d_workspace{nullptr};
  hipMalloc(&d_workspace, workspace_bytes);

  int image_bytes = 1 * 3 * height * width * sizeof(float);

//  float array[1][height][width][3];
  float* array = new float[3 * height * width];
  float* h_output = new float[3 * height * width];

  float* d_input{nullptr};
  hipMalloc(&d_input, image_bytes);
  hipMemset(d_input, 0, image_bytes);

  float* d_output{nullptr};
  hipMalloc(&d_output, image_bytes);
  hipMemset(d_output, 0, image_bytes);  
  
  const float kernel_template[3][3] = {
    {1, 1, 1},
    {1, -8, 1},
    {1, 1, 1}
  };

  float h_kernel[3][3][3][3];
  for (int kernel = 0; kernel < 3; ++kernel) {
    for (int channel = 0; channel <3; ++channel) {
      for (int row = 0; row < 3; ++row) {
        for (int column = 0; column < 3; ++column) {
          h_kernel[kernel][channel][row][column] = kernel_template[row][column];
        }
      }
    }
  }
  std::cout << "Sizeof of kernel " << sizeof(h_kernel) << std::endl;
  float* d_kernel{nullptr};
  hipMalloc(&d_kernel, sizeof(float) * 3 * 3 * 3 * 3);
  hipMemcpy(d_kernel, h_kernel, sizeof(h_kernel), hipMemcpyHostToDevice);

  float time;
  hipEvent_t start, stop;

  std::cout << "Transfer size: " << image_bytes * 1e-6 << " MB\n";
  HANDLE_ERROR( hipEventCreate(&start) );
  HANDLE_ERROR( hipEventCreate(&stop) );
  HANDLE_ERROR( hipEventRecord(start, 0) );

  int copy_cpu_cuda_repeat_times = 10000;
  for (int i = 0; i < copy_cpu_cuda_repeat_times; i++) {
    hipMemcpy(d_input, array, image_bytes, hipMemcpyHostToDevice);
  }

  HANDLE_ERROR( hipEventRecord(stop, 0) );
  HANDLE_ERROR( hipEventSynchronize(stop) );
  HANDLE_ERROR( hipEventElapsedTime(&time, start, stop) );

  delete[] array;

  std::cout << "[GPU]Run " << copy_cpu_cuda_repeat_times 
            << " cpu to cuda, run " << image_bytes * 1e-6 * copy_cpu_cuda_repeat_times / time * 1.0 << " GB/s\n";

  time = 0;
  start = 0;
  stop = 0;

  HANDLE_ERROR( hipEventCreate(&start) );
  HANDLE_ERROR( hipEventCreate(&stop) );
  HANDLE_ERROR( hipEventRecord(start, 0) );

  int conv_loop_times = 10000;
  for (int i = 0; i < conv_loop_times; i++) {
    const float alpha = 1, beta = 0;
    checkCUDNN(cudnnConvolutionForward(cudnn,
                                    &alpha,
                                    input_descriptor,
                                    d_input,
                                    kernel_descriptor,
                                    d_kernel,
                                    convolution_descriptor,
                                    convolution_algorithm,
                                    d_workspace,
                                    workspace_bytes,
                                    &beta,
                                    output_descriptor,
                                    d_output));
  }

  HANDLE_ERROR( hipEventRecord(stop, 0) );
  HANDLE_ERROR( hipEventSynchronize(stop) );
  HANDLE_ERROR( hipEventElapsedTime(&time, start, stop) );

  std::cout << "[GPU]Run " << conv_loop_times 
            << " convolutions, run " << image_bytes * 1e-6 * conv_loop_times / time * 1.0  << " GB/s\n";

  time = 0;
  start = 0;
  stop = 0;

  // Memory copy overhead
  HANDLE_ERROR( hipEventCreate(&start) );
  HANDLE_ERROR( hipEventCreate(&stop) );
  HANDLE_ERROR( hipEventRecord(start, 0) );

  int copy_cuda_cpu_repeat_times = 10000;
  for (int i = 0;  i < copy_cuda_cpu_repeat_times; i++) {
    hipMemcpy(h_output, d_output, image_bytes, hipMemcpyDeviceToHost);
    // Do something with h_output ...
  }

  HANDLE_ERROR( hipEventRecord(stop, 0) );
  HANDLE_ERROR( hipEventSynchronize(stop) );
  HANDLE_ERROR( hipEventElapsedTime(&time, start, stop) );
  std::cout << "[GPU]Run " << copy_cuda_cpu_repeat_times 
            << " cuda to cpu, run " << image_bytes * 1e-6 * copy_cuda_cpu_repeat_times / time * 1.0  << " GB/s\n";

  delete[] h_output;
  hipFree(d_kernel);
  hipFree(d_input);
  hipFree(d_output);
  hipFree(d_workspace);

  cudnnDestroyTensorDescriptor(input_descriptor);
  cudnnDestroyTensorDescriptor(output_descriptor);
  cudnnDestroyFilterDescriptor(kernel_descriptor);
  cudnnDestroyConvolutionDescriptor(convolution_descriptor);

  cudnnDestroy(cudnn);

}
